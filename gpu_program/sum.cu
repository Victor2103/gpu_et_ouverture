
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// code=sum && nvcc -o $code.o $code.cu && ./$code.o
// code=sum && nvcc -arch=sm_35 -o $code.o $code.cu && ./$code.o

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void d_sum(int* d_a, int* d_b, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
		atomicAdd(d_b, d_a[index]);
}

void h_sum(int* a, int* b, int size) {
    *b = 0;
    for (int i = 0; i < size; i++) {
        *b += a[i];
    }
}

void sum(int* a, int* b, int size) {
    int* deviceCount = (int*) malloc(sizeof(int));
    hipGetDeviceCount(deviceCount);
    if (*deviceCount == 0) {
        h_sum(a, b, size);
    } else {
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        printf(
            "threadsPerBlock=%d, blocksPerGrid=%d\n",
            threadsPerBlock, blocksPerGrid
        );

        int* d_a;
        int* d_b;

        gpuErrchk(hipMalloc((void**) &d_a, size * sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_b, sizeof(int)));

        gpuErrchk(hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice));

        d_sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, size);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipMemcpy(b, d_b, sizeof(int), hipMemcpyDeviceToHost));

        gpuErrchk(hipFree(d_a));
        gpuErrchk(hipFree(d_b));
    }
    printf("deviceCount = %d\n", *deviceCount);
    free(deviceCount);
}

void initialize(int* v, int size) {
    for (int i = 0; i < size; i++) {
        v[i] = 1;
    }
}

void print(int* v, int size) {
    printf("[");
    for (int i = 0; i < size - 1; i++) {
        printf("%d, ", v[i]);
    }
    if (size != 0) printf("%d", v[size - 1]);
    printf("]\n");
}

int main(int argc, char** argv) {
    int SIZE = 32;
    if (argc > 1) SIZE = (int) atoi(argv[1]);
    printf("SIZE=%d\n", SIZE);

    int* a = (int*) malloc(SIZE * sizeof(int));
    int* b = (int*) malloc(sizeof(int));

    initialize(a, SIZE);

    printf("a = ");
    print(a, SIZE);

    sum(a, b, SIZE);

    printf("b = ");
    print(b, 1);

    free(a);
    free(b);

    return 0;
}